#include "hip/hip_runtime.h"
#include "mass_kernel.hpp"
#include <cstdint>

template <typename T, int ndofs>
static __global__ void _mass_apply(std::int32_t num_elements, const T* xe, const T* phi,
                                   const T* detJ, T* ye) {
  __shared__ T _phi[ndofs][ndofs];
  __shared__ T _xe[ndofs];
  __shared__ T _xq[ndofs];

  if (threadIdx.x < ndofs) {
#pragma unroll
    for (int j = 0; j < ndofs; j++) {
      _phi[threadIdx.x][j] = phi[j * ndofs + threadIdx.x];
    }

    for (int block = blockIdx.x; block < num_elements; block += gridDim.x) {
      int id = block * ndofs + threadIdx.x;
      _xe[threadIdx.x] = xe[id];

      // Evaluate coefficients at quadrature points
      T wq = 0.;
#pragma unroll
      for (int j = 0; j < ndofs; j++)
        wq += _xe[j] * _phi[threadIdx.x][j];

      _xq[threadIdx.x] = detJ[id] * wq;

      T yi = 0;
#pragma unroll
      for (int iq = 0; iq < ndofs; iq++)
        yi += _xq[iq] * _phi[iq][threadIdx.x];

      ye[id] = yi;
    }
  }
}

template <typename T, int ndofs>
void mass_apply(int num_elements, const T* xe, const T* phi, const T* detJ, T* ye) {
  int block_size = 32 * ((ndofs + 32 - 1) / 32);
  const int num_blocks = num_elements / 8;
  dim3 dimBlock(block_size);
  dim3 dimGrid(num_blocks);
  _mass_apply<T, ndofs><<<dimGrid, dimBlock>>>(num_elements, xe, phi, detJ, ye);
}

//-----------------------------------------------------------------------------
template void mass_apply<double, 8>(int Ne, const double* xe, const double* phi,
                                    const double* detJ, double* ye);
template void mass_apply<double, 27>(int Ne, const double* xe, const double* phi,
                                     const double* detJ, double* ye);
template void mass_apply<double, 64>(int Ne, const double* xe, const double* phi,
                                     const double* detJ, double* ye);
//-----------------------------------------------------------------------------